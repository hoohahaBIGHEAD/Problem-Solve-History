﻿#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>


#define THREAD_NUM 128  // CUDA kernel should generate 128 threads per block

__global__ void stencil1D(int* a, int* c, int vec_size, int radius) {
    //a
    int* temp = (int*)malloc(vec_size * sizeof(int));
    temp[threadIdx.x] = a[threadIdx.x];
    int result = 0;
    /*
    if (threadIdx.x + offset < 0)
    {
        for (offset = 0; offset <= radius; offset++)
            result += temp[threadIdx.x + offset];
    }
    else if (vec_size < threadIdx.x + radius)
    {
        for (offset = 0; threadIdx.x + radius <= vec_size; offset++)
            result += temp[threadIdx.x + offset];
    }
    else
    {
        for (offset = -radius; offset <= radius; offset++)
            result += temp[threadIdx.x + offset];
    }
    */

    __syncthreads();

    
    for (int offset = -radius; offset <= radius; offset++) {
        if (threadIdx.x + offset < 0 || vec_size < threadIdx.x + offset)
            continue;
        else
            result += a[threadIdx.x + offset];
    }
    c[threadIdx.x] = result;

}

void vector_init(int* x, int size)
{
    int i;
    for (i = 0; i < size; i++) {
        x[i] = i;
    }
}



int main(void)
{
    int N, Radius, * a, * c, * d_a, * d_c;
    printf("vector size, and radius of stencil :");
    scanf("%d %d", &N, &Radius); // get the size of vectors as a keyboard input

  //b
    // Alloc space for device copies
    hipMalloc((void**)&d_a, N);
    hipMalloc((void**)&d_c, N);

    a = (int*)malloc(N * sizeof(int)); vector_init(a, N);
    c = (int*)malloc(N * sizeof(int));

    //c
    // Copy to device
    hipMemcpy(d_a, a, N, hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, N, hipMemcpyHostToDevice);

   //d stencil1D
    //stencil1D << <N, N >> > (d_a, d_c, 20, 4);
    stencil1D<<<1, N>>>(d_a, d_c, N, Radius);
    //e
    // Copy result back to host
    hipMemcpy(c, d_c, N, hipMemcpyDeviceToHost);

    for (int i=0;i<N;i++) printf("a[%d]=%d , c[%d]=%d\n",i,a[i],i,c[i]);       
    free(a); free(c); hipFree(d_a); hipFree(d_c);
    return 0;





}
